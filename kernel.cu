#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <windows.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <chrono>
#include <cmath>

using namespace std;
using namespace std::chrono;

struct header1
{
	char bfType[2];
	int bfSize;
	char bfReserved1[2];
	char bfReserved2[2];
	int bfOffBits;
};

struct header2
{
	int biSize;
	int biWidth;
	int biHeight;
	char biPlanes[2];
	char biBitCount[2];
	int biCompression;
	int biSizeImage;
	int biXpelsPerMeter;
	int biYpelsPerMeter;
	int biCrlUses;
	int biCrlImportant;
};

#define tablica(i,j) tablica[(i)*kol+(j)]
#define mask(i,j) mask[(i)*kol+(j)]

//Funkcja odczytujaca naglowek pliku
//==============================================================================================================================================================================
void WczytanieNagl(header1 &FileHeader11, header2 &FileHeader12, int &polozenie1)
{
	ifstream plik1("image.bmp", ios::binary);
	if (!plik1)
	{
		cout << "Blad otwarcia pliku. Koniec programu." << endl;
		exit(0);
	}

	plik1.read(reinterpret_cast<char *>(&FileHeader11.bfType), 2);
	plik1.read(reinterpret_cast<char *>(&FileHeader11.bfSize), 4);
	plik1.read(reinterpret_cast<char *>(&FileHeader11.bfReserved1), 2);
	plik1.read(reinterpret_cast<char *>(&FileHeader11.bfReserved2), 2);
	plik1.read(reinterpret_cast<char *>(&FileHeader11.bfOffBits), 4);


	plik1.read(reinterpret_cast<char *>(&FileHeader12.biSize), 4);
	plik1.read(reinterpret_cast<char *>(&FileHeader12.biWidth), 4);
	plik1.read(reinterpret_cast<char *>(&FileHeader12.biHeight), 4);
	plik1.read(reinterpret_cast<char *>(&FileHeader12.biPlanes), 2);
	plik1.read(reinterpret_cast<char *>(&FileHeader12.biBitCount), 2);
	plik1.read(reinterpret_cast<char *>(&FileHeader12.biCompression), 4);
	plik1.read(reinterpret_cast<char *>(&FileHeader12.biSizeImage), 4);
	plik1.read(reinterpret_cast<char *>(&FileHeader12.biXpelsPerMeter), 4);
	plik1.read(reinterpret_cast<char *>(&FileHeader12.biYpelsPerMeter), 4);
	plik1.read(reinterpret_cast<char *>(&FileHeader12.biCrlUses), 4);
	plik1.read(reinterpret_cast<char *>(&FileHeader12.biCrlImportant), 4);

	polozenie1 = plik1.tellg();
	//cout << polozenie1 << endl;

	//cout << FileHeader12.biWidth << endl;
	//cout << FileHeader12.biHeight << endl;
	//cout << FileHeader12.biSize << endl;

	plik1.close();

	//cout << FileHeader11.bfType << endl << FileHeader11.bfSize << endl << FileHeader11.bfReserved1 << endl << FileHeader11.bfReserved2 << endl << FileHeader11.bfOffBits << endl;
	//cout << FileHeader12.biSize << endl << FileHeader12.biWidth << endl << FileHeader12.biHeight << endl << FileHeader12.biPlanes << endl << FileHeader12.biBitCount << endl << FileHeader12.biCompression << endl << FileHeader12.biSizeImage << endl << FileHeader12.biXpelsPerMeter << endl << FileHeader12.biYpelsPerMeter << endl << FileHeader12.biCrlUses << endl << FileHeader12.biCrlImportant << endl;
}
//==============================================================================================================================================================================



//Funkcja odczytujaca dane o obrazie
//==============================================================================================================================================================================
void WczytanieObr(int polozenie1, int kol, int wier, int *tablica)
{

	ifstream plik1("image.bmp", ios::binary);
	if (!plik1)
	{
		cout << "Blad otwarcia pliku. Koniec programu." << endl;
		exit(0);
	}
	plik1.seekg(polozenie1);

	//Wype³nienie tablicy zerami w celu unikniêcia b³êdów sczytywania danych
	for (int i = 0; i < wier; i++)
	{
		for (int j = 0; j < kol; j++)
		{
			tablica(i, j) = 0;
		}
	}

	for (int i = 0; i < wier; i++)
	{
		for (int j = 0; j < kol; j++)
		{
			plik1.read(reinterpret_cast<char *>(&tablica(i,j)), 1);
		}
	}
	
	plik1.close();
}
//==============================================================================================================================================================================


//Funkcja zapisujaca dane do nowego pliku z zastosowaniem filtru dolnoprzepustowego na CPU
//==============================================================================================================================================================================
void FiltrDolCPU(header1 FileHeader11, header2 FileHeader12, int kol, int wier, int *tablica)
{
	ofstream plik("image_low_pass_filter_CPU.bmp", ios::binary);
	if (!plik)
	{
		cout << "Blad utworzenia pliku. Koniec programu." << endl;
		exit(0);
	}

	//Zapis naglowka
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfType), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfSize), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfReserved1), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfReserved2), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfOffBits), 4);


	plik.write(reinterpret_cast<char *>(&FileHeader12.biSize), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biWidth), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biHeight), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biPlanes), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biBitCount), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biCompression), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biSizeImage), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biXpelsPerMeter), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biYpelsPerMeter), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biCrlUses), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biCrlImportant), 4);

	//Zapis danych o obrazie
	auto start = high_resolution_clock::now();
	for (int i = 0; i < wier; i++)
	{
		for (int j = 0; j < kol; j++)
		{
			int mask = tablica(i, j);
			if(i == 0 || i == wier - 1 || j < 3 || j > kol - 4)
			plik.write(reinterpret_cast<char *>(&tablica(i,j)), 1);
			else
			{
				int mask = tablica(i, j) + tablica(i+1, j+3) + tablica(i-1, j-3) + tablica(i+1, j-3) + tablica(i-1, j+3) + tablica(i+1, j) + tablica(i-1, j) + tablica(i, j+3) + tablica(i, j-3);
				mask = (int)(mask / 9);
				mask = abs(mask);
				plik.write(reinterpret_cast<char *>(&mask), 1);
			}
		}
	}
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	cout << "Czas nakladania maski filtru dolnoprzepustowego na CPU: " << duration.count() << endl << endl;

	plik.close();
}
//==============================================================================================================================================================================


//Funkcja zapisujaca dane do nowego pliku z zastosowaniem filtru gornoprzepustowego na CPU
//==============================================================================================================================================================================
void FiltrGorCPU(header1 FileHeader11, header2 FileHeader12, int kol, int wier, int *tablica)
{
	ofstream plik("image_high_pass_filter_CPU.bmp", ios::binary);
	if (!plik)
	{
		cout << "Blad utworzenia pliku. Koniec programu." << endl;
		exit(0);
	}

	//Zapis naglowka
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfType), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfSize), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfReserved1), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfReserved2), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfOffBits), 4);


	plik.write(reinterpret_cast<char *>(&FileHeader12.biSize), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biWidth), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biHeight), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biPlanes), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biBitCount), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biCompression), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biSizeImage), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biXpelsPerMeter), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biYpelsPerMeter), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biCrlUses), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biCrlImportant), 4);

	//Zapis danych o obrazie
	auto start = high_resolution_clock::now();
	for (int i = 0; i < wier; i++)
	{
		for (int j = 0; j < kol; j++)
		{
			int mask = tablica(i, j);
			if (i == 0 || i == wier - 1 || j < 3 || j > kol - 4)
				plik.write(reinterpret_cast<char *>(&tablica(i, j)), 1);
			else
			{
				//int mask = 5*tablica(i, j) - tablica(i + 1, j + 3) - tablica(i - 1, j - 3) - tablica(i + 1, j - 3) - tablica(i - 1, j + 3) - tablica(i + 1, j) - tablica(i - 1, j) - tablica(i, j + 3) - tablica(i, j - 3);
				int mask = 5 * tablica(i, j) - tablica(i + 1, j) - tablica(i - 1, j) - tablica(i, j + 3) - tablica(i, j - 3);
				mask = (int)(mask / 1);
				mask = abs(mask);
				plik.write(reinterpret_cast<char *>(&mask), 1);
			}
		}
	}
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	cout << "Czas nakladania maski filtru gornoprzepustowego na CPU: " << duration.count() << endl << endl;

	plik.close();
}
//==============================================================================================================================================================================


//KERNEL->filtr dolnoprzepustowy
//==============================================================================================================================================================================
__global__ void FiltrDol(const int *tablica, int *mask, int kol, int wier)
{
	int ko = threadIdx.y + blockIdx.y * blockDim.y;
	int wi = threadIdx.x + blockIdx.x * blockDim.x;
	if (ko < kol && wi < wier)
	{
		if (wi == 0 || wi == wier - 1 || ko < 3 || ko > kol - 4)
			mask(wi, ko) = tablica(wi, ko);
		else
		{
			mask(wi, ko) = tablica(wi, ko) + tablica(wi + 1, ko + 3) + tablica(wi - 1, ko - 3) + tablica(wi + 1, ko - 3) + tablica(wi - 1, ko + 3) + tablica(wi + 1, ko) + tablica(wi - 1, ko) + tablica(wi, ko + 3) + tablica(wi, ko - 3);
			mask(wi, ko) = (int)(mask(wi, ko) / 9);
			mask(wi, ko) = abs(mask(wi, ko));
		}
	}
}
//==============================================================================================================================================================================


//KERNEL->filtr gornoprzepustowy
//==============================================================================================================================================================================
__global__ void FiltrGor(const int *tablica, int *mask, int kol, int wier)
{
	int ko = threadIdx.y + blockIdx.y * blockDim.y;
	int wi = threadIdx.x + blockIdx.x * blockDim.x;
	if (ko < kol && wi < wier)
	{
		if (wi == 0 || wi == wier - 1 || ko < 3 || ko > kol - 4)
			mask(wi, ko) = tablica(wi, ko);
		else
		{
			//int mask(wi, ko) = 5*tablica(wi, ko) - tablica(wi + 1, ko + 3) - tablica(wi - 1, ko - 3) - tablica(wi + 1, ko - 3) - tablica(wi - 1, ko + 3) - tablica(wi + 1, ko) - tablica(wi - 1, ko) - tablica(wi, ko + 3) - tablica(wi, ko - 3);
			mask(wi, ko) = 5 * tablica(wi, ko) - tablica(wi + 1, ko) - tablica(wi - 1, ko) - tablica(wi, ko + 3) - tablica(wi, ko - 3);
			mask(wi, ko) = (int)(mask(wi, ko) / 1);
			mask(wi, ko) = abs(mask(wi, ko));
		}
	}
}
//==============================================================================================================================================================================


//Funkcja zapisujaca dane do nowego pliku z zastosowaniem filtru dolnoprzepustowego na GPU
//==============================================================================================================================================================================
void FiltrDolGPU(header1 FileHeader11, header2 FileHeader12, int kol, int wier, int *tablica)
{
	ofstream plik("image_low_pass_filter_GPU.bmp", ios::binary);
	if (!plik)
	{
		cout << "Blad utworzenia pliku. Koniec programu." << endl;
		exit(0);
	}

	//Zapis naglowka
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfType), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfSize), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfReserved1), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfReserved2), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfOffBits), 4);


	plik.write(reinterpret_cast<char *>(&FileHeader12.biSize), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biWidth), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biHeight), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biPlanes), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biBitCount), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biCompression), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biSizeImage), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biXpelsPerMeter), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biYpelsPerMeter), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biCrlUses), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biCrlImportant), 4);

	int *mask;
	int *tab_cuda = 0, *mask_cuda = 0;

	mask = (int *)malloc(sizeof(*mask)*wier*kol);
	hipMalloc((void**)&tab_cuda, wier*kol * sizeof(*tab_cuda));
	hipMalloc((void**)&mask_cuda, wier*kol * sizeof(*mask_cuda));
	hipMemcpy(tab_cuda, tablica, sizeof(*tab_cuda)*wier*kol, hipMemcpyHostToDevice);

	dim3 blockDim(10, 10);
	dim3 gridDim(1000, 1000);

	auto start = high_resolution_clock::now();
	//==============================================================================================================================================================================
	FiltrDol << < gridDim, blockDim >> > (tab_cuda, mask_cuda, kol, wier);
	//==============================================================================================================================================================================
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	cout << "Czas nakladania maski filtru dolnoprzepustowego na GPU: " << duration.count() << endl << endl;

	hipMemcpy(mask, mask_cuda, wier*kol * sizeof(*mask), hipMemcpyDeviceToHost);

	for (int i = 0; i < wier; i++)
	{
		for (int j = 0; j < kol; j++)
		{
			plik.write(reinterpret_cast<char *>(&mask(i, j)), 1);
		}
	}

	free(mask);
	hipFree(tab_cuda);
	hipFree(mask_cuda);

	plik.close();
}
//==============================================================================================================================================================================


//Funkcja zapisujaca dane do nowego pliku z zastosowaniem filtru gornoprzepustowego na GPU
//==============================================================================================================================================================================
void FiltrGorGPU(header1 FileHeader11, header2 FileHeader12, int kol, int wier, int *tablica)
{
	ofstream plik("image_high_pass_filter_GPU.bmp", ios::binary);
	if (!plik)
	{
		cout << "Blad utworzenia pliku. Koniec programu." << endl;
		exit(0);
	}

	//Zapis naglowka
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfType), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfSize), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfReserved1), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfReserved2), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader11.bfOffBits), 4);


	plik.write(reinterpret_cast<char *>(&FileHeader12.biSize), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biWidth), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biHeight), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biPlanes), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biBitCount), 2);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biCompression), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biSizeImage), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biXpelsPerMeter), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biYpelsPerMeter), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biCrlUses), 4);
	plik.write(reinterpret_cast<char *>(&FileHeader12.biCrlImportant), 4);

	int *mask;
	int *tab_cuda = 0, *mask_cuda = 0;

	mask = (int *)malloc(sizeof(*mask)*wier*kol);
	hipMalloc((void**)&tab_cuda, wier*kol * sizeof(*tab_cuda));
	hipMalloc((void**)&mask_cuda, wier*kol * sizeof(*mask_cuda));
	hipMemcpy(tab_cuda, tablica, sizeof(*tab_cuda)*wier*kol, hipMemcpyHostToDevice);

	dim3 blockDim(10, 10);
	dim3 gridDim(1000, 1000);

	auto start = high_resolution_clock::now();
	//==============================================================================================================================================================================
	FiltrGor << < gridDim, blockDim >> > (tab_cuda, mask_cuda, kol, wier);
	//==============================================================================================================================================================================
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	cout << "Czas nakladania maski filtru gornoprzepustowego na GPU: " << duration.count() << endl << endl;

	hipMemcpy(mask, mask_cuda, wier*kol * sizeof(*mask), hipMemcpyDeviceToHost);

	for (int i = 0; i < wier; i++)
	{
		for (int j = 0; j < kol; j++)
		{
			plik.write(reinterpret_cast<char *>(&mask(i, j)), 1);
		}
	}

	free(mask);
	hipFree(tab_cuda);
	hipFree(mask_cuda);

	plik.close();
}
//==============================================================================================================================================================================


int main()
{
	header1 FileHeader11;
	header2 FileHeader12;
	int polozenie1;
	WczytanieNagl(FileHeader11, FileHeader12, polozenie1);

	int kol = FileHeader12.biWidth * 3;
	if(kol%4 != 0)
		kol = FileHeader12.biWidth * 3 + (4 - (FileHeader12.biWidth * 3) % 4);
	int wier = FileHeader12.biHeight;

	int *tablica;
	tablica = (int *)malloc(sizeof(*tablica)*wier*kol);
	WczytanieObr(polozenie1, kol, wier, tablica);

	FiltrDolCPU(FileHeader11, FileHeader12, kol, wier, tablica);
	FiltrDolGPU(FileHeader11, FileHeader12, kol, wier, tablica);
	FiltrGorCPU(FileHeader11, FileHeader12, kol, wier, tablica);
	FiltrGorGPU(FileHeader11, FileHeader12, kol, wier, tablica);

	free(tablica);
	
	return(0);
}
